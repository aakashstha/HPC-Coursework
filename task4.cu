#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>

#include "lodepng.h"

/*
    Example:-
     To Compile = nvcc task4.cu -o xyz lodepng.cpp
     To Run = ./xyz

	 The output of this task-4 program is a png image name = "WOW_Image.png".
*/
__global__ void boxBlur(unsigned char *gpu_imageOuput, unsigned char *gpu_imageInput, int width, int height);
__device__ void RGB_SumDivide(int j, int d, unsigned char *gpu_imageOuput, unsigned char *gpu_imageInput, int r1, int r2, int r3, int r4, int r5, int r6, int r7, int r8, int r9);

int main(int argc, char **argv)
{
	unsigned int error, encError, width, height;
	unsigned char *image;
	const char *filename = "Data/image.png";
	const char *newFileName = "WOW_Image.png";

	error = lodepng_decode32_file(&image, &width, &height, filename);
	if (error)
		printf("error %u: %s\n", error, lodepng_error_text(error));

	const int ARRAY_SIZE = width * height * 4;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(unsigned char);

	unsigned char host_imageInput[ARRAY_SIZE * 4];
	unsigned char host_imageOutput[ARRAY_SIZE * 4];

	for (int i = 0; i < ARRAY_SIZE; i++)
		host_imageInput[i] = image[i];

	// declare GPU memory pointers
	unsigned char *d_in, *d_out;

	// allocate GPU memory
	hipMalloc((void **)&d_in, ARRAY_BYTES);
	hipMalloc((void **)&d_out, ARRAY_BYTES);

	hipMemcpy(d_in, host_imageInput, ARRAY_BYTES, hipMemcpyHostToDevice);

	// launch the kernel
	boxBlur<<<height, width>>>(d_out, d_in, width, height);

	// copy back the result array to the CPU
	hipMemcpy(host_imageOutput, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

	encError = lodepng_encode32_file(newFileName, host_imageOutput, width, height);
	if (encError)
		printf("error %u: %s\n", error, lodepng_error_text(encError));

	printf("Image width = %d and height = %d.\nOutput image name = %s\n", width, height, newFileName);
	hipFree(d_in);
	hipFree(d_out);

	return 0;
}

__global__ void boxBlur(unsigned char *gpu_imageOuput, unsigned char *gpu_imageInput, int width, int height)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int pixel = idx * 4;

	int i = pixel;
	int row = 4 * blockDim.x;
	int full = row * gridDim.x;

	unsigned int r1 = 0, r2 = 0, r3 = 0, r4 = 0, r5 = 0, r6 = 0, r7 = 0, r8 = 0, r9 = 0;

	if (i == 0) //Left topmost corner pixel
	{
		r1 = i, r2 = i + 4, r3 = row, r4 = row + 4;
		RGB_SumDivide(i, 4, gpu_imageOuput, gpu_imageInput, r1, r2, r3, r4, r5, r6, r7, r8, r9);
	}
	else if (i == (row - 4)) //(i==4)  Right topmost corner pixel
	{
		r1 = i - 4, r2 = i, r3 = row * 2 - 4 - 4, r4 = row * 2 - 4;
		RGB_SumDivide(i, 4, gpu_imageOuput, gpu_imageInput, r1, r2, r3, r4, r5, r6, r7, r8, r9);
	}
	else if (i == (row * height) - row) //(i==20)  Left bottom-most corner pixel
	{
		r1 = full - (row * 2), r2 = full - (row * 2) + 4, r3 = i, r4 = i + 4;
		RGB_SumDivide(i, 4, gpu_imageOuput, gpu_imageInput, r1, r2, r3, r4, r5, r6, r7, r8, r9);
	}
	else if (i == (row * height - 4)) //i==24  Right bottom-most corner pixel
	{
		r1 = full - (row * 2) - (4 * 2), r2 = full - (row * 2) - 4, r3 = i - 4, r4 = i;
		RGB_SumDivide(i, 4, gpu_imageOuput, gpu_imageInput, r1, r2, r3, r4, r5, r6, r7, r8, r9);
	}
	else if (i < (row)) // Top Edge Row
	{
		r1 = i - 4, r2 = i, r3 = i + 4, r4 = row + (i - 4), r5 = row + i, r6 = row + i + 4;
		RGB_SumDivide(i, 6, gpu_imageOuput, gpu_imageInput, r1, r2, r3, r4, r5, r6, r7, r8, r9);
	}
	else if (i == (row * (int)(i / row))) // Left Edge Row
	{
		r1 = i - row, r2 = i - row + 4, r3 = i, r4 = i + 4, r5 = i + row, r6 = i + row + 4;
		RGB_SumDivide(i, 6, gpu_imageOuput, gpu_imageInput, r1, r2, r3, r4, r5, r6, r7, r8, r9);
	}

	else if (i == row * (int)((i + row) / row) - 4) // Rigth Edge Row
	{
		r1 = i - row - 4, r2 = i - row, r3 = i - 4, r4 = i, r5 = i + row - 4, r6 = i + row;
		RGB_SumDivide(i, 6, gpu_imageOuput, gpu_imageInput, r1, r2, r3, r4, r5, r6, r7, r8, r9);
	}
	else if (i > (row * height) - row) // Bottom Edge Row
	{
		r1 = i - row - 4, r2 = i - row, r3 = i - row + 4, r4 = i - 4, r5 = i, r6 = i + 4;
		RGB_SumDivide(i, 6, gpu_imageOuput, gpu_imageInput, r1, r2, r3, r4, r5, r6, r7, r8, r9);
	}
	else // All Center Pixels
	{
		r1 = i - row + 4, r2 = i + 4 - row + 4, r3 = i + 4 + 4 - row + 4, r4 = i - 4, r5 = i, r6 = i + 4, r7 = i + row - 4, r8 = i + row, r9 = i + row + 4;
		RGB_SumDivide(i, 9, gpu_imageOuput, gpu_imageInput, r1, r2, r3, r4, r5, r6, r7, r8, r9);
	}
}

__device__ void RGB_SumDivide(int j, int d, unsigned char *gpu_imageOuput, unsigned char *gpu_imageInput, int r1, int r2, int r3, int r4, int r5, int r6, int r7, int r8, int r9)
{
	unsigned int r = 0, g = 0, b = 0, t = 0;
	if (d == 4)
	{
		r = gpu_imageInput[r1] + gpu_imageInput[r2] + gpu_imageInput[r3] + gpu_imageInput[r4];
		g = gpu_imageInput[r1 + 1] + gpu_imageInput[r2 + 1] + gpu_imageInput[r3 + 1] + gpu_imageInput[r4 + 1];
		b = gpu_imageInput[r1 + 2] + gpu_imageInput[r2 + 2] + gpu_imageInput[r3 + 2] + gpu_imageInput[r4 + 2];
	}
	else if (d == 6)
	{
		r = gpu_imageInput[r1] + gpu_imageInput[r2] + gpu_imageInput[r3] + gpu_imageInput[r4] + gpu_imageInput[r5] + gpu_imageInput[r6];
		g = gpu_imageInput[r1 + 1] + gpu_imageInput[r2 + 1] + gpu_imageInput[r3 + 1] + gpu_imageInput[r4 + 1] + gpu_imageInput[r5 + 1] + gpu_imageInput[r6 + 1];
		b = gpu_imageInput[r1 + 2] + gpu_imageInput[r2 + 2] + gpu_imageInput[r3 + 2] + gpu_imageInput[r4 + 2] + gpu_imageInput[r5 + 2] + gpu_imageInput[r6 + 2];
	}
	else if (d == 9)
	{
		r = gpu_imageInput[r1] + gpu_imageInput[r2] + gpu_imageInput[r3] + gpu_imageInput[r4] + gpu_imageInput[r5] + gpu_imageInput[r6] + gpu_imageInput[r7] + gpu_imageInput[r8] + gpu_imageInput[r9];
		g = gpu_imageInput[r1 + 1] + gpu_imageInput[r2 + 1] + gpu_imageInput[r3 + 1] + gpu_imageInput[r4 + 1] + gpu_imageInput[r5 + 1] + gpu_imageInput[r6 + 1] + gpu_imageInput[r7 + 1] + gpu_imageInput[r8 + 1] + gpu_imageInput[r9 + 1];
		b = gpu_imageInput[r1 + 2] + gpu_imageInput[r2 + 2] + gpu_imageInput[r3 + 2] + gpu_imageInput[r4 + 2] + gpu_imageInput[r5 + 2] + gpu_imageInput[r6 + 2] + gpu_imageInput[r7 + 2] + gpu_imageInput[r8 + 2] + gpu_imageInput[r9 + 2];
	}

	t = gpu_imageInput[j + 3];
	gpu_imageOuput[j] = r / d;
	gpu_imageOuput[j + 1] = g / d;
	gpu_imageOuput[j + 2] = b / d;
	gpu_imageOuput[j + 3] = t;
}
