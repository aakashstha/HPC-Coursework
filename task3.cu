#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>

		//__global__ --> GPU function which can be launched by many blocks and threads
		//__device__ --> GPU function or variables
		//__host__ --> CPU function or variables

		/*
	A Z 0 0 = CCBDWY2244
	M W 2 3 = OKNZTV4071
	P O 8 8 = RNQRLN1634
	Z Z 9 9 = CXBDWY2745
	*/

		__device__ char *
		CudaCrypt(char *rawPassword)
{
	char *newPassword = (char *)malloc(sizeof(char) * 11);

	newPassword[0] = rawPassword[0] + 2;
	newPassword[1] = rawPassword[0] - 2;
	newPassword[2] = rawPassword[0] + 1;
	newPassword[3] = rawPassword[1] + 3;
	newPassword[4] = rawPassword[1] - 3;
	newPassword[5] = rawPassword[1] - 1;
	newPassword[6] = rawPassword[2] + 2;
	newPassword[7] = rawPassword[2] - 2;
	newPassword[8] = rawPassword[3] + 4;
	newPassword[9] = rawPassword[3] - 4;
	newPassword[10] = '\0';

	for (int i = 0; i < 10; i++)
	{
		if (i >= 0 && i < 6)
		{ //checking all lower case letter limits
			if (newPassword[i] > 90)
			{
				newPassword[i] = (newPassword[i] - 90) + 65;
			}
			else if (newPassword[i] < 65)
			{
				newPassword[i] = (65 - newPassword[i]) + 65;
			}
		}
		else
		{ //checking number section
			if (newPassword[i] > 57)
			{
				newPassword[i] = (newPassword[i] - 57) + 48;
			}
			else if (newPassword[i] < 48)
			{
				newPassword[i] = (48 - newPassword[i]) + 48;
			}
		}
	}
	return newPassword;
}

__device__ int compareTwoEncryption(char *first, char *second)
{
	for (int i = 0; i < 10; i++)
	{
		if (first[i] != second[i])
		{
			return 0;
		}
	}
	return 1;
}

__global__ void crack(char *alphabet, char *numbers, char *encrypted_password)
{

	char genRawPass[4];

	genRawPass[0] = alphabet[blockIdx.x];
	genRawPass[1] = alphabet[blockIdx.y];

	genRawPass[2] = numbers[threadIdx.x];
	genRawPass[3] = numbers[threadIdx.y];

	char *generateEncryptPassword = CudaCrypt(genRawPass);

	int flag = compareTwoEncryption(generateEncryptPassword, encrypted_password);
	if (flag)
	{
		printf("Password Cracked %s encryption was %s\n", genRawPass, encrypted_password);
	}
	else
	{
		printf("Password not cracked %s encryption was %s\n", genRawPass, encrypted_password);
	}

	//printf("%c %c %c %c = %s\n", genRawPass[0], genRawPass[1], genRawPass[2], genRawPass[3], CudaCrypt(genRawPass));
}

int main(int argc, char **argv)
{
	char cpuEncrypted_password[10] = "CCBDWY2244";
	char *gpuEncrypted_password;
	char cpuAlphabet[26] = {'A', 'B', 'C', 'D', 'E', 'F', 'G', 'H', 'I', 'J', 'K', 'L', 'M', 'N', 'O', 'P', 'Q', 'R', 'S', 'T', 'U', 'V', 'W', 'X', 'Y', 'Z'};
	char cpuNumbers[26] = {'0', '1', '2', '3', '4', '5', '6', '7', '8', '9'};

	char *gpuAlphabet;
	hipMalloc((void **)&gpuAlphabet, sizeof(char) * 26);
	hipMalloc((void **)&gpuEncrypted_password, sizeof(cpuEncrypted_password));

	hipMemcpy(gpuAlphabet, cpuAlphabet, sizeof(char) * 26, hipMemcpyHostToDevice);
	hipMemcpy(gpuEncrypted_password, cpuEncrypted_password, sizeof(cpuEncrypted_password), hipMemcpyHostToDevice);

	char *gpuNumbers;
	hipMalloc((void **)&gpuNumbers, sizeof(char) * 26);
	hipMemcpy(gpuNumbers, cpuNumbers, sizeof(char) * 26, hipMemcpyHostToDevice);

	crack<<<dim3(26, 26, 1), dim3(10, 10, 1)>>>(gpuAlphabet, gpuNumbers, gpuEncrypted_password);
	hipDeviceSynchronize();
	return 0;
}