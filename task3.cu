#include "hip/hip_runtime.h"
% % cu
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>

		/*
	__global__ --> GPU function which can be launched by many blocks and threads
	__device__ --> GPU function or variables
	__host__ --> CPU function or variables


	## some of the encrypted password you can try!
	// AZ00 = CCBDWY2244
	// MW23 = OKNZTV4071
	// PO88 = RNQRLN1634
	// ZZ99 = CXBDWY2745
*/

		// To Encrypt All the Characters and Numbers
		__device__ char *
		CudaCrypt(char *rawPassword)
{
	char *newPassword = (char *)malloc(sizeof(char) * 11);

	newPassword[0] = rawPassword[0] + 2;
	newPassword[1] = rawPassword[0] - 2;
	newPassword[2] = rawPassword[0] + 1;
	newPassword[3] = rawPassword[1] + 3;
	newPassword[4] = rawPassword[1] - 3;
	newPassword[5] = rawPassword[1] - 1;
	newPassword[6] = rawPassword[2] + 2;
	newPassword[7] = rawPassword[2] - 2;
	newPassword[8] = rawPassword[3] + 4;
	newPassword[9] = rawPassword[3] - 4;
	newPassword[10] = '\0';

	for (int i = 0; i < 10; i++)
	{
		if (i >= 0 && i < 6)
		{ //checking all lower case letter limits
			if (newPassword[i] > 90)
			{
				newPassword[i] = (newPassword[i] - 90) + 65;
			}
			else if (newPassword[i] < 65)
			{
				newPassword[i] = (65 - newPassword[i]) + 65;
			}
		}
		else
		{ //checking number section
			if (newPassword[i] > 57)
			{
				newPassword[i] = (newPassword[i] - 57) + 48;
			}
			else if (newPassword[i] < 48)
			{
				newPassword[i] = (48 - newPassword[i]) + 48;
			}
		}
	}
	return newPassword;
}

// To check all the encrypted password match with each other or not
__device__ int compareTwoEncryption(char *encrypted, char *generated)
{
	for (int i = 0; i < 10; i++)
	{
		if (encrypted[i] != generated[i])
			return 0;
	}
	return 1;
}

__global__ void crack(char *alphabet, char *numbers, char *encrypted_password)
{
	char genRawPass[4];

	genRawPass[0] = alphabet[blockIdx.x];
	genRawPass[1] = alphabet[blockIdx.y];

	genRawPass[2] = numbers[threadIdx.x];
	genRawPass[3] = numbers[threadIdx.y];

	char *generateEncryptPassword = CudaCrypt(genRawPass);
	int flag = compareTwoEncryption(encrypted_password, generateEncryptPassword);
	if (flag)
	{
		encrypted_password[0] = genRawPass[0];
		encrypted_password[1] = genRawPass[1];

		encrypted_password[2] = genRawPass[2];
		encrypted_password[3] = genRawPass[3];
		printf("Password Cracked %s encryption was %s\n", genRawPass, encrypted_password);
	}

	//printf("%c %c %c %c = %s\n", genRawPass[0], genRawPass[1], genRawPass[2], genRawPass[3], CudaCrypt(genRawPass));
}

int main(int argc, char **argv)
{
	char cpuAlphabet[26] = {'A', 'B', 'C', 'D', 'E', 'F', 'G', 'H', 'I', 'J', 'K', 'L', 'M', 'N', 'O', 'P', 'Q', 'R', 'S', 'T', 'U', 'V', 'W', 'X', 'Y', 'Z'};
	char cpuNumbers[26] = {'0', '1', '2', '3', '4', '5', '6', '7', '8', '9'};

	char *gpuAlphabet, *gpuNumbers;
	char encrypted_password[11] = "CCBDWY2244";
	char *encrypted_password2;
	char *gpuEncrypted_password;

	hipMalloc((void **)&gpuAlphabet, sizeof(char) * 26);
	hipMalloc((void **)&gpuNumbers, sizeof(char) * 26);
	hipMalloc((void **)&gpuEncrypted_password, sizeof(char) * 11);

	hipMemcpy(gpuAlphabet, cpuAlphabet, sizeof(char) * 26, hipMemcpyHostToDevice);
	hipMemcpy(gpuNumbers, cpuNumbers, sizeof(char) * 26, hipMemcpyHostToDevice);
	hipMemcpy(gpuEncrypted_password, encrypted_password, sizeof(char) * 11, hipMemcpyHostToDevice);

	crack<<<dim3(26, 26, 1), dim3(10, 10, 1)> > >(gpuAlphabet, gpuNumbers, gpuEncrypted_password);
	hipDeviceSynchronize();

	//hipMemcpy(encrypted_password2, gpuEncrypted_password, sizeof(char) * 11, hipMemcpyDeviceToHost);

	hipFree(gpuAlphabet);
	hipFree(gpuNumbers);
	hipFree(gpuEncrypted_password);

	//printf("%s \n", encrypted_password2);

	return 0;
}