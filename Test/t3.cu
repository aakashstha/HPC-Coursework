#include "hip/hip_runtime.h"
// To run on Google Colab
%%cu
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>

/*
	__global__ --> GPU function which can be launched by many blocks and threads
	__device__ --> GPU function or variables
	__host__ --> CPU function or variables

	While running this task3 on Google Colab just uncomment the first statement written %%cu
	and it will run perfectly fine.


	## some of the encrypted password you can try!
	// AZ00 = CCBDWY2244
	// MW23 = OKNZTV4071
	// PO88 = RNQRLN1634
	// ZZ99 = CXBDWY2745
*/
__device__ char * CudaCrypt(char *rawPassword);
__device__ int compareTwoEncryption(char *encrypted, char *generated);
__global__ void crack(char *alphabet, char *numbers, char *encrypted_password, char *crack_password);

int main(int argc, char **argv)
{
	
	char cpuAlphabet[26] = {'A', 'B', 'C', 'D', 'E', 'F', 'G', 'H', 'I', 'J', 'K', 'L', 'M', 'N', 'O', 'P', 'Q', 'R', 'S', 'T', 'U', 'V', 'W', 'X', 'Y', 'Z'};
	char cpuNumbers[26] = {'0', '1', '2', '3', '4', '5', '6', '7', '8', '9'};

	char *gpuAlphabet, *gpuNumbers;
	char encrypted_password[11] = "CXBDWY2745";
	char *gpuCrack_password;
	char *cpuCracked_password = (char *)malloc(sizeof(char) * 11);
	char *gpuEncrypted_password;

	hipMalloc((void **)&gpuAlphabet, sizeof(char) * 26);
	hipMalloc((void **)&gpuNumbers, sizeof(char) * 26);
	hipMalloc((void **)&gpuEncrypted_password, sizeof(char) * 11);
	hipMalloc((void **)&gpuCrack_password, sizeof(char) * 11);

	hipMemcpy(gpuAlphabet, cpuAlphabet, sizeof(char) * 26, hipMemcpyHostToDevice);
	hipMemcpy(gpuNumbers, cpuNumbers, sizeof(char) * 26, hipMemcpyHostToDevice);
	hipMemcpy(gpuEncrypted_password, encrypted_password, sizeof(char) * 11, hipMemcpyHostToDevice);

	crack<<<dim3(26, 26, 1), dim3(10, 10, 1)>>>(gpuAlphabet, gpuNumbers, gpuEncrypted_password, gpuCrack_password);
	hipDeviceSynchronize();

	hipMemcpy(cpuCracked_password, gpuCrack_password, sizeof(char) * 11, hipMemcpyDeviceToHost);

	hipFree(gpuAlphabet);
	hipFree(gpuNumbers);
	hipFree(gpuEncrypted_password);

	printf("PASSWORD CRACKED!!\nReceived encryption was and generated encryption is %s.\nYour password is = %s\n", encrypted_password, cpuCracked_password);

	//printf("%s \n", encrypted_password2);

	return 0;
}

__global__ void crack(char *alphabet, char *numbers, char *encrypted_password, char *crack_password)
{
	char genRawPass[4];

	genRawPass[0] = alphabet[blockIdx.x];
	genRawPass[1] = alphabet[blockIdx.y];

	genRawPass[2] = numbers[threadIdx.x];
	genRawPass[3] = numbers[threadIdx.y];

	char *generateEncryptPassword = CudaCrypt(genRawPass);
	int flag = compareTwoEncryption(encrypted_password, generateEncryptPassword);
	if (flag)
	{
		crack_password[0] = genRawPass[0];
		crack_password[1] = genRawPass[1];

		crack_password[2] = genRawPass[2];
		crack_password[3] = genRawPass[3];
		// printf("PASSWORD CRACKED!!\nReceived encryption was %s and generated encryption is %s.\nYour password is = %s\n", encrypted_password, generateEncryptPassword, genRawPass);
	}

	//printf("%c %c %c %c = %s\n", genRawPass[0], genRawPass[1], genRawPass[2], genRawPass[3], CudaCrypt(genRawPass));
}

// To Encrypt All the Characters and Numbers
__device__ char * CudaCrypt(char *rawPassword)
{
	char *newPassword = (char *)malloc(sizeof(char) * 11);

	newPassword[0] = rawPassword[0] + 2;
	newPassword[1] = rawPassword[0] - 2;
	newPassword[2] = rawPassword[0] + 1;
	newPassword[3] = rawPassword[1] + 3;
	newPassword[4] = rawPassword[1] - 3;
	newPassword[5] = rawPassword[1] - 1;
	newPassword[6] = rawPassword[2] + 2;
	newPassword[7] = rawPassword[2] - 2;
	newPassword[8] = rawPassword[3] + 4;
	newPassword[9] = rawPassword[3] - 4;
	newPassword[10] = '\0';

	for (int i = 0; i < 10; i++)
	{
		if (i >= 0 && i < 6)
		{ //checking all lower case letter limits
			if (newPassword[i] > 90)
			{
				newPassword[i] = (newPassword[i] - 90) + 65;
			}
			else if (newPassword[i] < 65)
			{
				newPassword[i] = (65 - newPassword[i]) + 65;
			}
		}
		else
		{ //checking number section
			if (newPassword[i] > 57)
			{
				newPassword[i] = (newPassword[i] - 57) + 48;
			}
			else if (newPassword[i] < 48)
			{
				newPassword[i] = (48 - newPassword[i]) + 48;
			}
		}
	}
	return newPassword;
}

// To check all the encrypted password match with each other or not
__device__ int compareTwoEncryption(char *encrypted, char *generated)
{
	for (int i = 0; i < 10; i++)
	{
		if (encrypted[i] != generated[i])
			return 0;
	}
	return 1;
}

